
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
__global__ void computeSquare(int *d_in, int *d_out) {
	int index = threadIdx.x;
	d_out[index] = d_in[index] * d_in[index];
 }
int main() {
	const int arr_size = 5;
	int h_in[arr_size] = { 1, 2, 3, 4, 5 };
	int h_out[arr_size];
	int arr_bytes = arr_size * sizeof(int);
	int *d_in, *d_out;
	hipMalloc((void **)&d_in, arr_bytes);
	hipMalloc((void **)&d_out, arr_bytes);
	hipMemcpy(d_in, h_in, arr_bytes, hipMemcpyHostToDevice);
	computeSquare <<<1, arr_size >>>(d_in, d_out);
	hipMemcpy(h_out, d_out, arr_bytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < 5; i++) {
		printf("%d ", h_out[i]);
	}
	system("pause");
	return 0;
}